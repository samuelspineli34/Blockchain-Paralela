#include "hip/hip_runtime.h"
#include "Block.h"
#include "sha256.cuh"
#include <iostream>
#include <hip/hip_runtime.h>

const int BLOCK_SIZE = 256;

__device__ void CalculateHash(uint32_t index, const char* prevHash, time_t time, const char* data, uint32_t nonce, char* hash) {
    // Concatenate inputs into a single string
    char buffer[256];  // Adjust size as needed
    snprintf(buffer, sizeof(buffer), "%u%s%ld%s%u", index, prevHash, time, data, nonce);

    // Call the SHA-256 function
    sha256(buffer, hash);
}

__global__ void mineBlockKernel(uint32_t* nonce, const char* target, uint32_t difficulty, char* hash, uint32_t index, const char* prevHash, time_t time, const char* data) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t localNonce = tid;
    char localHash[65];  // Assuming sha256 produces a 64-character hash

    while (true) {
        // Calculate hash for current nonce
        CalculateHash(index, prevHash, time, data, localNonce, localHash);

        // Check if the hash meets the difficulty target
        bool valid = true;
        for (uint32_t i = 0; i < difficulty; ++i) {
            if (localHash[i] != '0') {
                valid = false;
                break;
            }
        }

        if (valid) {
            // Found a valid nonce, store it and break
            if (atomicMin(nonce, localNonce) > localNonce) {
                // Copy the valid hash to the global memory
                memcpy(hash, localHash, 64);
                hash[64] = '\0';
            }
            break;
        }

        // Increment local nonce and continue
        localNonce += gridDim.x * blockDim.x;
    }
}

void Block::MineBlock(uint32_t nDifficulty) {
    // Allocate GPU memory
    uint32_t* devNonce;
    char* devTarget;
    char* devHash;

    hipMalloc((void**)&devNonce, sizeof(uint32_t));
    hipMalloc((void**)&devTarget, nDifficulty + 1);
    hipMalloc((void**)&devHash, 65);  // Assuming 64-character hash plus null terminator

    // Initialize nonce and target on the device
    uint32_t maxNonce = UINT32_MAX;
    hipMemcpy(devNonce, &maxNonce, sizeof(uint32_t), hipMemcpyHostToDevice);

    char* target = new char[nDifficulty + 1];
    memset(target, '0', nDifficulty);
    target[nDifficulty] = '\0';
    hipMemcpy(devTarget, target, nDifficulty + 1, hipMemcpyHostToDevice);
    delete[] target;

    // Launch kernel
    mineBlockKernel<<<1, BLOCK_SIZE>>>(devNonce, devTarget, nDifficulty, devHash, _nIndex, sPrevHash.c_str(), _tTime, _sData.c_str());

    // Wait for GPU to finish
    hipDeviceSynchronize();

    // Copy results back to host
    char hostHash[65];
    hipMemcpy(&_nNonce, devNonce, sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipMemcpy(hostHash, devHash, 65, hipMemcpyDeviceToHost);
    sHash = std::string(hostHash);

    // Cleanup
    hipFree(devNonce);
    hipFree(devTarget);
    hipFree(devHash);

    std::cout << "Block mined: " << sHash << std::endl;
}
